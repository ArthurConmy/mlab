
#include <hip/hip_runtime.h>
constexpr size_t BLOCK_SIZE = 512;
__global__ void sum_shared(int32_t* in, int64_t size, int32_t* out) { 
  __shared__ int32_t buf[BLOCK_SIZE];
  const size_t offset = BLOCK_SIZE * blockIdx.x;
  const size_t i = threadIdx.x;
  for (size_t i = 0; i < BLOCK_SIZE; i++) {
    const size_t in_idx = offset + i;
    buf[i] = in_idx < size ? in[in_idx] : 0;
  }
  __syncthreads();
  for (size_t gap = 256; gap > 0; gap /= 2) {
    if (i < gap) {
      buf[i] += buf[i + gap];
    }
    __syncthreads();
  }
  if (i == 0) {
    out[blockIdx.x] = buf[0];
  }
}
