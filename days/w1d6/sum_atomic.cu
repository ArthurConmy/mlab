
#include <hip/hip_runtime.h>
__global__ void sumAtomic(float *values, float *sum, int inputSize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < inputSize) {
        atomicAdd(sum, values[idx]);
    }
}

